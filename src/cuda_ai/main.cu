#include "hip/hip_runtime.h"
/*
	2048 Game for HPC Group Assignment
	Liron Mizrahi 708810
	Jason Chalom 711985
	
	Cuda Ai
*/

#define CUDA True //this is to use the same library functions

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stack>

#include "../helper/helper.h"

// Includes CUDA
#include <hip/hip_runtime.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define heading "CUDA Dave Ai for playing 2048 using matrix approach"
#define results_header "children,nodes,time,win rate,flops"
#define results_location "../results/results_cuda1_ai.csv"

/* Global variables */
#define app_name "Cuda 2048 AI - DaveAi"
    
int board_size = 4;
bool use_rnd = false;
int max_depth = -1;
int max_num_nodes = 10000;
bool save_to_file = false;
bool print_output = false;
bool print_path = false;
bool save_csv = false;
string initial_state_path = "";
string filepath = "./results/cuda _ai";
bool DEBUG = false;
float time_limit = -1.0;

#define DIM 1024
#define warp 32

// Auto-Verification Code
bool testResult = true;

using namespace std;

/* Function Headers */
int main(int argc, char *argv[]);

/* device functions */
__global__ void buildTree(Node** device_arr, int* device_num_sub_tree_nodes, int* board_size, hiprandState_t* rnd_states);
__global__ void init_rnd(unsigned int seed, hiprandState_t* states, int* device_num_sub_tree_nodes);
__device__ bool cuda_add_new_number(GameState *currentGame, hiprandState_t* states, int* device_num_sub_tree_nodes);

__device__ void cuda_process_action(GameState *currentGame, int action, int* boardSize);
__device__ void cuda_process_left(GameState *currentGame, int* boardSize);
__device__ void cuda_process_right(GameState *currentGame, int* boardSize);
__device__ void cuda_process_up(GameState *currentGame, int* boardSize);
__device__ void cuda_process_down(GameState *currentGame, int* boardSize);

/* host functions */
void run_AI();
void serialBuildTree(Tree* tree, int leaf_node_limit, Node** host_arr);
void serialGenerateChidlren(Node* currentNode, Tree* tree, Node** host_arr);
Node* createHostTreeArray(Tree* tree, int num_host_leaves, int num_sub_tree_nodes);
    
void process_args(int argc, char *argv[]);
void halt_execution_cuda(string);

int main(int argc, char *argv[])
{
	//some basic setup
    print_cmd_heading(app_name);
    
    if (argc == 1)
    {
        print_usage(argc, argv);
        halt_execution_cuda("");
    }
    
    if(use_rnd)
        srand(time(NULL));
    else
        srand(10000);
    
    process_args(argc, argv);
    run_AI();

	hipDeviceReset();
    printf("%s completed, returned %s\n",
           heading,
           testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void run_AI()
{
    if(print_output)
        printf("Init...\n");
    
    float time_taken = 0.0;
    
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);    
    float start_epoch = sdkGetTimerValue(&timer);
    
    GameState* initial_state = new GameState(board_size);
	add_new_number(initial_state);

	Tree* tree = new Tree(initial_state);
    stack<Node*> tracker;
    
    int num_host_leaves = 1024; //todo: dynamic calcs
    int num_sub_tree_nodes = 1024;
    
    size_t height = num_host_leaves+4;
    size_t width = num_sub_tree_nodes*sizeof(Node);
    size_t nodeArrSize = height*width;
    
    if(print_output)
        printf("Allocate host arr...\n");
    Node* host_arr[height]; //= (Node**)malloc(sizeof(Node*)*num_host_leaves+4);
    for(int i = 0; i < height; i++)
        host_arr[i] = (Node *)malloc(width);
    
    if(print_output)
        printf("Building initial tree...\n");
    serialBuildTree(tree, num_host_leaves, host_arr);
    
    if(print_output)
        printf("Move host array to device...\n");
    
    // device variables
    Node** device_arr;
    int* device_num_sub_tree_nodes;
    int* device_board_size;
    size_t devPitch;
    
    // checkCudaErrors(hipMalloc((void**)&device_arr, nodeArrSize));
    checkCudaErrors(hipMallocPitch((void**)&device_board_size, &devPitch, width, height));
    checkCudaErrors(hipMalloc((void**)&device_num_sub_tree_nodes, sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_board_size, sizeof(int)));
    
    checkCudaErrors(hipMemcpy2D(device_board_size, devPitch, &host_arr, nodeArrSize, width, height, hipMemcpyHostToDevice));

    printf("4...\n");
    checkCudaErrors(hipMemcpy(device_num_sub_tree_nodes, &max_num_nodes, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_board_size, &board_size, sizeof(int), hipMemcpyHostToDevice));
    
    dim3 dimBlock( warp, warp, 1 );
    dim3 dimGrid( board_size*warp, board_size*warp );
    
    if(print_output)
        printf("Start buildTree kernel...\n");
    
    //hiprand stuff
    unsigned long int seed;
    if(use_rnd)
        seed = time(NULL);
    else
        seed = 10000;
    
    hiprandState_t* states;
    hipMalloc((void**) &states, warp*warp* sizeof(hiprandState_t)); //N*sizeof
    
    init_rnd<<<dimGrid, dimBlock>>>(seed, states, device_num_sub_tree_nodes);
	buildTree<<<dimGrid, dimBlock>>>(device_arr, device_num_sub_tree_nodes, device_board_size, states);
    
    if(print_output)
        printf("Copy results back to host...\n\n");
    // hipMemcpy(host_arr, device_arr, nodeArrSize, hipMemcpyDeviceToHost);
    checkCudaErrors(hipMemcpy2D(host_arr, devPitch, device_arr, nodeArrSize, width, height, hipMemcpyDeviceToHost));
    
    float end_epoch = sdkGetTimerValue(&timer);
    time_taken = end_epoch-start_epoch;
    
    if(print_path)
    {
        // print_solution(tree);
    }
    
    if(print_output)
    {
//         printf("board_size: %i, num_nodes: %d, max_depth: %d, sols: %d, leaves: %d, stats: %f\n", board_size, tree->num_nodes, tree->max_depth, tree->num_solutions, tree->num_leaves, ((double)tree->num_solutions/(double)tree->num_leaves));
        
//         if(tree->optimal2048)
//             printf("min_depth: %d time_taken: %f\n", tree->optimal2048->depth, time_taken);
    }

    
    if(save_to_file)
    {
        printf("Save optimal path to file...\n");
//         if (save_csv)
//             filepath.append(".csv");
//         else
//             filepath.append(".txt");
                            
//         save_solution_to_file(tree, time_taken, filepath, save_csv);
    }
    
    /* cleanup */
    sdkDeleteTimer(&timer);
	checkCudaErrors(hipFree(device_arr));
    checkCudaErrors(hipFree(device_num_sub_tree_nodes));
    checkCudaErrors(hipFree(device_board_size));
}

__global__ void buildTree(Node** device_arr, int* device_num_sub_tree_nodes, int* board_size, hiprandState_t* rnd_states)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = (*device_num_sub_tree_nodes)*y+x;
    
    int curr_node = 0;
    
    while(curr_node < *device_num_sub_tree_nodes-4)
    {
        Node* currentNode = &device_arr[x][curr_node];
        
        for (int i = 0; i < 4; i++)
        {
            GameState* newState = new GameState(*board_size);
            newState->copy(currentNode->current_state);

            cuda_process_action(newState, i, board_size);

            if(!determine_2048(currentNode->current_state) && !compare_game_states(currentNode->current_state, newState))
            {
                bool fullBoard = !cuda_add_new_number(newState, rnd_states, device_num_sub_tree_nodes);
                if(!fullBoard)
                {
                    int currentDepth = currentNode->depth + 1;
                    // if(tree->max_depth < currentDepth)
                    //     tree->max_depth = currentDepth;

                    // currentNode->children[i] = new Node(currentNode, newState, currentDepth);
                    Node newNode(currentNode, newState, currentDepth);
                    device_arr[x][curr_node+i+1] = newNode;
                    currentNode->children[i] = &device_arr[x][curr_node+i+1];
                    // tree->num_nodes++;

                    currentNode->hasChildren = true;
                }
                else
                {
                    currentNode->children[i] = nullptr;
                }
            }
            else
            {
                currentNode->children[i] = nullptr;
            }
        }
        
        curr_node++;
    }
}

__global__ void init_rnd(unsigned int seed, hiprandState_t* states, int* device_num_sub_tree_nodes) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = (*device_num_sub_tree_nodes)*y+x;
    hiprand_init(seed, idx, 0, &states[blockIdx.x]);
}

__device__ bool cuda_add_new_number(GameState *currentGame, hiprandState_t* states, int* device_num_sub_tree_nodes)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = (*device_num_sub_tree_nodes)*y+x;
    
    int rand_row = hiprand(&states[idx]) % currentGame->boardSize;
    int rand_col = hiprand(&states[idx]) % currentGame->boardSize;

	if(checkBoardEmptySlot(currentGame))
	{
		while(currentGame->currentBoard[rand_row][rand_col] != 0)
		{
            rand_row = hiprand(&states[idx]) % currentGame->boardSize;
            rand_col = hiprand(&states[idx]) % currentGame->boardSize;
		}

		currentGame->currentBoard[rand_row][rand_col] = 2;
		return true;
	}
	return false;
}

__device__ void cuda_process_action(GameState *currentGame, int action, int* boardSize)
{
	if (action == 0)
	{
		cuda_process_left(currentGame, boardSize);
	}
	else if (action == 1)
	{
		cuda_process_right(currentGame, boardSize);
	}
	else if (action == 2)
	{
		cuda_process_up(currentGame, boardSize);
	}
	else if (action == 3)
	{
		cuda_process_down(currentGame, boardSize);
	}
}

__device__ void cuda_process_left(GameState *currentGame, int* boardSize)
{
    bool* modified = (bool*)malloc(sizeof(bool)*(*boardSize));
	for (int i = 0; i < *boardSize; ++i)
	{
		for (int p = 0; p < *boardSize; ++p)
		{
			modified[p] = false;
		}

		for (int j = 1; j < *boardSize; ++j)
		{
			if (currentGame->currentBoard[i][j] != 0)
			{
				int t = j;
				while(t > 0 && currentGame->currentBoard[i][t-1] == 0)
				{
					currentGame->currentBoard[i][t-1] = currentGame->currentBoard[i][t];
					currentGame->currentBoard[i][t] = 0;
					t--;
				}

				if (t == 0)
				{
					t++;
				}

				if (currentGame->currentBoard[i][t-1] == currentGame->currentBoard[i][t] &&  modified[t-1] == false)
				{
					currentGame->currentBoard[i][t-1] += currentGame->currentBoard[i][t];
					modified[t-1] = true;
					currentGame->score += currentGame->currentBoard[i][t-1];
					currentGame->currentBoard[i][t] = 0;
				}
			}
			
		}
	}
    free(modified);
}

__device__ void cuda_process_right(GameState *currentGame, int* boardSize)
{
    bool* modified = (bool*)malloc(sizeof(bool)*(*boardSize));
	for (int i = 0; i < *boardSize; ++i)
	{          
		for (int p = 0; p < *boardSize; ++p)
		{
			modified[p] = false;
		}

		for (int j = *boardSize - 2; j > -1; --j)
		{
			if (currentGame->currentBoard[i][j] != 0)
			{
				int t = j;
				while(t < *boardSize - 1 && currentGame->currentBoard[i][t+1] == 0)
				{
					currentGame->currentBoard[i][t+1] = currentGame->currentBoard[i][t];
					currentGame->currentBoard[i][t] = 0;
					t++;
				}

				if (t == *boardSize - 1)
				{
					t--;
				}

				if (currentGame->currentBoard[i][t+1] == currentGame->currentBoard[i][t] && modified[t+1] == false)
				{
					currentGame->currentBoard[i][t+1] += currentGame->currentBoard[i][t];
					modified[t+1] = true;
					currentGame->score += currentGame->currentBoard[i][t+1];
					currentGame->currentBoard[i][t] = 0;
				}
			}
		}
	}
    free(modified);
}

__device__ void cuda_process_up(GameState *currentGame, int* boardSize)
{
    bool* modified = (bool*)malloc(sizeof(bool)*(*boardSize));   
	for (int j = 0; j < *boardSize; ++j)
	{          
		for (int p = 0; p < *boardSize; ++p)
		{
			modified[p] = false;
		}

		for (int i = 1; i < *boardSize; ++i)
		{
			if (currentGame->currentBoard[i][j] != 0)
			{
				int t = i;
				while(t > 0 && currentGame->currentBoard[t-1][j] == 0)
				{
					currentGame->currentBoard[t-1][j] = currentGame->currentBoard[t][j];
					currentGame->currentBoard[t][j] = 0;
					t--;
				}

				if (t == 0)
				{
					t++;
				}

				if (currentGame->currentBoard[t-1][j] == currentGame->currentBoard[t][j] &&  modified[t-1] == false)
				{
					currentGame->currentBoard[t-1][j] += currentGame->currentBoard[t][j];
					modified[t+1] = true;
					currentGame->score += currentGame->currentBoard[i][t-1];
					currentGame->currentBoard[t][j] = 0;
				}
			}
		}
	}
    free(modified);
}

__device__ void cuda_process_down(GameState *currentGame, int* boardSize)
{
    bool* modified = (bool*)malloc(sizeof(bool)*(*boardSize));   
	for (int j = 0; j < *boardSize; ++j)
	{      
		for (int p = 0; p < *boardSize; ++p)
		{
			modified[p] = false;
		}

		for (int i = *boardSize - 2; i > -1; --i)
		{
			if (currentGame->currentBoard[i][j] != 0)
			{
				int t = i;
				while(t < *boardSize - 1 && currentGame->currentBoard[t+1][j] == 0)
				{
					currentGame->currentBoard[t+1][j] = currentGame->currentBoard[t][j];
					currentGame->currentBoard[t][j] = 0;
					t++;
				}

				if (t == *boardSize - 1)
				{
					t--;
				}

				if (currentGame->currentBoard[t+1][j] == currentGame->currentBoard[t][j] && modified[t+1] == false)
				{
					currentGame->currentBoard[t+1][j] += currentGame->currentBoard[t][j];
					modified[t+1] = true;
					currentGame->score += currentGame->currentBoard[t+1][j];
					currentGame->currentBoard[t][j] = 0;
				}
			}
		}
	}
    free(modified);
}

void serialBuildTree(Tree* tree, int leaf_node_limit, Node** host_arr)
{
    //todo: fix this not working correctly
	stack<Node*> tracker;
	tracker.push(tree->root);

	while(!tracker.empty() && !shouldLimit(tree, leaf_node_limit))
	{
		Node* currentNode = tracker.top();
        tracker.pop();
    
		if(currentNode)
		{
			serialGenerateChidlren(currentNode, tree, host_arr);
            
            for (int i = 3; i > -1; --i)
            {
                tracker.push(currentNode->children[i]);
            }
		}
        
        if(DEBUG)
        {
            printf("%lui\n", tracker.size());
        }
    }
}

void serialGenerateChidlren(Node* currentNode, Tree* tree, Node** host_arr)
{
	for (int i = 0; i < 4; i++)
	{
        GameState* newState = new GameState(tree->BOARD_SIZE);
		newState->copy(currentNode->current_state);
        
		process_action(newState, i);

        if(!determine_2048(currentNode->current_state) && !compare_game_states(currentNode->current_state, newState))
        {
            bool fullBoard = !add_new_number(newState);
            if(!fullBoard)
            {
                int currentDepth = currentNode->depth + 1;
                if(tree->max_depth < currentDepth)
                    tree->max_depth = currentDepth;
                
                currentNode->children[i] = new Node(currentNode, newState, currentDepth);
                tree->num_nodes++;

                currentNode->hasChildren = true;
            }
            else
            {
                currentNode->children[i] = nullptr;
            }
        }
        else
        {
            currentNode->children[i] = nullptr;
        }

        if(determine_2048(currentNode->current_state)) //win and shortest path
        {
            if(tree->optimal2048)
            {
                if(currentNode->depth < tree->optimal2048->depth) 
                    tree->optimal2048 = currentNode;
            }
            else
                tree->optimal2048 = currentNode;
            
            tree->num_solutions++;
        }

        if(determine_2048(currentNode->current_state) || compare_game_states(currentNode->current_state, newState)) 
        {
            tree->num_leaves++;
        }
        
        if(!determine_2048(currentNode->current_state) && !compare_game_states(currentNode->current_state, newState)) 
        {
            host_arr[tree->num_cutoff_states][0] = *currentNode;
            tree->num_cutoff_states++;
        }
	}
    
    if(DEBUG)
    {
        printf("%d, %d\n", tree->num_nodes, tree->max_depth);
        print_board(currentNode->current_state);
    }
}

void process_args(int argc, char *argv[])
{
    for (int i = 1; i < argc; i++)
    {
        string str = string(argv[i]);
        if(contains_string(str, "board_size"))
        {
            board_size = atoi(str.substr(str.find('=') + 1).c_str());
            if(board_size < 2)
            {
                print_usage(argc, argv);
                halt_execution_cuda("\nError: board_size must be grater than 1.");
            }
        }
           
        if(contains_string(str, "use_rnd"))
        {
            use_rnd = true;
        }
           
        if(contains_string(str, "max_depth"))
        {
            max_depth = atoi(str.substr(str.find('=') + 1).c_str());
            if(max_depth < 2)
            {
                print_usage(argc, argv);
                halt_execution_cuda("\nError: max_depth must be grater than 1.");
            }
        }
           
        if(contains_string(str, "max_num_nodes"))
        {
            max_num_nodes = atoi(str.substr(str.find('=') + 1).c_str());
            if(max_num_nodes < 2)
            {
                print_usage(argc, argv);
                halt_execution_cuda("\nError: max_num_nodes must be grater than 1.");
            }
        }
           
        if(contains_string(str, "save_to_file"))
        {
            save_to_file = true;
        }
           
        if(contains_string(str, "print_output"))
        {
            print_output = true;
        }
           
        if(contains_string(str, "save_csv"))
        {
            save_csv = true;
        }
           
        if(contains_string(str, "filepath"))
        {
            filepath = str.substr(str.find('=') + 1);
        }
                                                                                                           
        if(contains_string(str, "print_path"))
        {
            print_path = true;
        }
           
        if(contains_string(str, "DEBUG"))
        {
            DEBUG = true;
        }
        
        if(contains_string(str, "usage"))
        {
            print_usage(argc, argv);
            halt_execution_cuda("");
        }

        if(contains_string(str, "time_limit"))
        {
            time_limit = atof(str.substr(str.find('=') + 1).c_str());
        }
    }
}

void halt_execution_cuda(string message="")
{
    hipDeviceReset();
	halt_execution(message);
}