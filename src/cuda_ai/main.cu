#include "hip/hip_runtime.h"
/*
	2048 Game for HPC Group Assignment
	Liron Mizrahi 708810
	Jason Chalom 711985
	
	Cuda Ai
*/

#define CUDA True //this is to use the same library functions
    
#include "../helper/helper.h"
#include "cuda_ai.h"
#include "cuda_2048.cpp"
#include "serial_tree_builder.cpp"

#define heading "CUDA Dave Ai for playing 2048 using matrix approach"
#define results_header "children,nodes,time,win rate,flops"
#define results_location "../results/results_cuda1_ai.csv"

/* Global variables */
#define app_name "Cuda 2048 AI - DaveAi"
    
int board_size = 4;
bool use_rnd = false;
int max_depth = -1;
int max_num_nodes = 10000;
bool save_to_file = false;
bool print_output = false;
bool print_path = false;
bool save_csv = false;
string initial_state_path = "";
string filepath = "./results/cuda _ai";
bool DEBUG = false;
float time_limit = -1.0;

int num_host_leaves = 32;//1024; //todo: dynamic calcs
int num_sub_tree_nodes = 1024; 

#define DIM 1024
#define warp 32

// Auto-Verification Code
bool testResult = true;

using namespace std;

int main(int argc, char *argv[])
{
	//some basic setup
    print_cmd_heading(app_name);
    
    if (argc == 1)
    {
        print_usage(argc, argv);
        halt_execution_cuda("");
    }

    process_args(argc, argv);
    
    if(use_rnd)
        srand(time(NULL));
    else
        srand(10000);
    

    run_AI();

	hipDeviceReset();
    printf("%s completed, returned %s\n",
           heading,
           testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void run_AI()
{
    if(print_output)
        printf("Init...\n");
    
    float time_taken = 0.0;
    
    Tree_Stats *tstats = new Tree_Stats;
    tstats->BOARD_SIZE = board_size;
    
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);    
    float start_epoch = sdkGetTimerValue(&timer);
    
    GameState* initial_state = new GameState(board_size);
	add_new_number(initial_state);

	Tree* tree = new Tree(initial_state);
    stack<Node*> tracker;
      
    size_t height = num_host_leaves; 
    size_t width = (num_sub_tree_nodes);
    size_t nodeArrSize = height*width *sizeof(Node);
                                                                                                                              
    if(print_output)
        printf("Allocate host arr...\n");
    Node* host_arr = (Node*)malloc(nodeArrSize);
    
    if(print_output)
        printf("Building initial tree...\n");

    std::stack<Node*> init_states;
    init_states = get_init_states(num_host_leaves); //gets all the cut off nodes for gpu
    
    for(unsigned int i = 0;i < height;i++)
    {
        host_arr[i*width] = *init_states.top();
        init_states.pop();
    }
    
    //update tree stats
    update_tree_stats(tstats, tree->root, tree->optimal2048, 0, tree->num_nodes, tree->max_depth, tree->num_solutions, tree->num_leaves, tree->num_cutoff_states);
    
    if(print_output)
        printf("Move host array to device...\n");
    
    // device variables
    Node* device_arr;
    Tree_Stats* device_tstats;
    int* device_num_sub_tree_nodes;
    
    int threadCounts[2] = {0, 0};
    calc_thread_count(threadCounts, height);
    dim3 dimBlock( threadCounts[0], threadCounts[1], 1 );
	dim3 dimGrid( 1, 1 );
    
    checkCudaErrors(hipMalloc((void**)&device_arr, nodeArrSize));
    checkCudaErrors(hipMalloc((void**)&device_tstats, sizeof(Tree_Stats)));
    checkCudaErrors(hipMalloc((void**)&device_num_sub_tree_nodes, sizeof(int)));
    checkCudaErrors(hipMemcpy(device_arr, host_arr, nodeArrSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_tstats, tstats, sizeof(Tree_Stats), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_num_sub_tree_nodes, &max_num_nodes, sizeof(int), hipMemcpyHostToDevice));
    
    if(print_output)
        printf("Start buildTree kernel...\n");
    
    //hiprand stuff
    unsigned long int seed;
    if(use_rnd)
        seed = time(NULL);
    else
        seed = 10000;
    
    hiprandState_t* states;
    hipMalloc((void**) &states, threadCounts[0]*threadCounts[1]*sizeof(hiprandState_t));
    
    init_rnd<<<dimGrid, dimBlock>>>(seed, states, device_num_sub_tree_nodes);
	buildTree<<<dimGrid, dimBlock>>>(device_arr, device_tstats, num_sub_tree_nodes, board_size, states, height, width, nodeArrSize);
    
    if(print_output)
        printf("Copy results back to host...\n\n");
    
    checkCudaErrors(hipMemcpy(host_arr, device_arr, nodeArrSize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(tstats, device_tstats, sizeof(Tree_Stats), hipMemcpyDeviceToHost));
    
    float end_epoch = sdkGetTimerValue(&timer);
    time_taken = end_epoch-start_epoch;
    
    if(print_path)
    {
        // print_solution(tree);
    }
    
    if(print_output)
    {
        printf("board_size: %i, num_nodes: %d, max_depth: %d, sols: %d, leaves: %d, stats: %f\n", board_size, tstats->num_nodes, tstats->max_depth, tstats->num_solutions, tstats->num_leaves, ((double)tstats->num_solutions/(double)tstats->num_leaves));
        
        if(tstats->optimal2048)
            printf("min_depth: %d time_taken: %f\n", tstats->optimal2048->depth, time_taken);
    }

    
    if(save_to_file)
    {
        printf("Save optimal path to file...\n");
        if (save_csv)
            filepath.append(".csv");
        else
            filepath.append(".txt");
                            
        save_solution_to_file(tree, time_taken, filepath, save_csv);
    }
    
    /* cleanup */
    sdkDeleteTimer(&timer);
	checkCudaErrors(hipFree(device_arr));
    checkCudaErrors(hipFree(device_tstats));
    checkCudaErrors(hipFree(device_num_sub_tree_nodes));
}

void calc_thread_count(int* threadCount, int height)
{
    if (height <= DIM)
    {
        threadCount[0] = height;
        threadCount[1] = 1;
    }
    else
    {
        double check = height / DIM;
        check = ceil(check);
        
        threadCount[0] = DIM;
        threadCount[1] = (int)check;
    }
    // printf("ThreadCount: %d, %d\n", threadCount[0], threadCount[1]);
}

__global__ void buildTree(Node* device_arr, Tree_Stats* device_tstats, int num_sub_tree_nodes, int board_size, hiprandState_t* rnd_states, size_t height, size_t width, size_t nodeArrSize)
{
    int idx = threadIdx.y * blockDim.x + threadIdx.x;
    if(threadIdx.x == 31)
    {
        printf("THREADIDX.X = %d\n", threadIdx.x);
        printf("THREADIDX.Y = %d\n", threadIdx.y);
        printf("IDX = %d\n", idx);
    }
        
    int curr_node = 0;
    
    // printf("Test %d\n", idx);
    while(curr_node < num_sub_tree_nodes) // curr_node < (height-4) && idx < num_sub_tree_nodes
    {
        int arr_idx = idx*width + curr_node;//curr_node+width*idx;
        
        // Node* currentNode = &device_arr[arr_idx];
        
        if(device_arr[arr_idx].isReal)
        {
            for (int i = 0; i < 4; i++)
            {
                // printf("bs: %d\n", this.boardSize);
                GameState newState = GameState(board_size);
                print_board(device_arr[arr_idx].current_state);
                newState.copy(device_arr[arr_idx].current_state); //why?

                cuda_process_action(&newState, i, board_size);
                
                if(!determine_2048(device_arr[arr_idx].current_state) && !compare_game_states(device_arr[arr_idx].current_state, &newState))
                {
                    bool fullBoard = !cuda_add_new_number(&newState, rnd_states, &num_sub_tree_nodes);
                    if(!fullBoard)
                    {
                        int currentDepth = device_arr[arr_idx].depth + 1;
                        // if(device_tstats.max_depth < currentDepth)
                        //     device_tstats.max_depth = currentDepth;

                        // device_arr[arr_idx].children[i] = new Node(device_arr[arr_idx], newState, currentDepth);
                        Node newNode(&device_arr[arr_idx], &newState, currentDepth);
                        int new_arr_idx = (4*arr_idx+(i+1));
                        device_arr[new_arr_idx] = newNode;
                        
                        device_arr[arr_idx].children[i] = &device_arr[new_arr_idx];
                        // tree.num_nodes++;
                        print_board(&newState);                    
                        // device_tstats.num_nodes++;
                        device_arr[arr_idx].hasChildren = true;
                    }
                    else
                    {
                        device_arr[arr_idx].children[i] = nullptr;
                        Node newNode = Node();
                        int new_arr_idx = 4*arr_idx+(i+1);
                        device_arr[new_arr_idx] = newNode;
                    }
                }
                else
                {
                    device_arr[arr_idx].children[i] = nullptr;
                    Node newNode = Node();
                    int new_arr_idx = 4*arr_idx+(i+1);
                    device_arr[new_arr_idx] = newNode;
                }
                
    //             if(determine_2048(currentNode.current_state)) //win and shortest path
    //             {
    //                 if(device_tstats->optimal2048)
    //                 {
    //                     if(currentNode->depth < device_tstats->optimal2048->depth) 
    //                         device_tstats->optimal2048 = currentNode;
    //                 }
    //                 else
    //                     device_tstats->optimal2048 = currentNode;

    //                 device_tstats->num_solutions++;
    //             }

    //             if(determine_2048(currentNode->current_state) || compare_game_states(currentNode->current_state, newState)) 
    //             {
    //                 device_tstats->num_leaves++;
    //             }

    //             if(!determine_2048(currentNode->current_state) && !compare_game_states(currentNode->current_state, newState)) 
    //             {
    //                 device_tstats->num_cutoff_states++;
    //             }  
            } 
        }
        curr_node++;
        // __syncthreads();
    }
    __syncthreads();
}

__global__ void init_rnd(unsigned int seed, hiprandState_t* states, int* device_num_sub_tree_nodes) {
    int idx = threadIdx.y * blockDim.x + threadIdx.x;
    
    hiprand_init(seed, idx, 0, &states[idx]);
}

//TODO:CMDLINE Stuff
void process_args(int argc, char *argv[])
{
    for (int i = 1; i < argc; i++)
    {
        string str = string(argv[i]);
        if(contains_string(str, "board_size"))
        {
            board_size = atoi(str.substr(str.find('=') + 1).c_str());
            if(board_size < 2)
            {
                print_usage(argc, argv);
                halt_execution_cuda("\nError: board_size must be grater than 1.");
            }
        }
           
        if(contains_string(str, "use_rnd"))
        {
            use_rnd = true;
        }
           
        if(contains_string(str, "max_depth"))
        {
            max_depth = atoi(str.substr(str.find('=') + 1).c_str());
            if(max_depth < 2)
            {
                print_usage(argc, argv);
                halt_execution_cuda("\nError: max_depth must be grater than 1.");
            }
        }
           
        if(contains_string(str, "max_num_nodes"))
        {
            max_num_nodes = atoi(str.substr(str.find('=') + 1).c_str());
            if(max_num_nodes < 2)
            {
                print_usage(argc, argv);
                halt_execution_cuda("\nError: max_num_nodes must be grater than 1.");
            }
        }
           
        if(contains_string(str, "save_to_file"))
        {
            save_to_file = true;
        }
           
        if(contains_string(str, "print_output"))
        {
            print_output = true;
        }
           
        if(contains_string(str, "save_csv"))
        {
            save_csv = true;
        }
           
        if(contains_string(str, "filepath"))
        {
            filepath = str.substr(str.find('=') + 1);
        }
                                                                                                           
        if(contains_string(str, "print_path"))
        {
            print_path = true;
        }
           
        if(contains_string(str, "DEBUG"))
        {
            DEBUG = true;
        }
        
        if(contains_string(str, "usage"))
        {
            print_usage(argc, argv);
            halt_execution_cuda("");
        }

        if(contains_string(str, "time_limit"))
        {
            time_limit = atof(str.substr(str.find('=') + 1).c_str());
        }
    }
}

void halt_execution_cuda(string message="")
{
    hipDeviceReset();
	halt_execution(message);
}